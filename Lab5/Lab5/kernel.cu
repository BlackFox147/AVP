#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include ""

#include <stdio.h>
#include <conio.h>
#include <Windows.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <ctime>
#include <intrin.h>
#include <omp.h>

const int n = 32;
//int N = 32;


void SumCPU(int *A, int *B, int N) {

	unsigned __int64 begin, end;

	srand((unsigned int)time(NULL));
	double time_s, time_f;
	time_s = omp_get_wtime();
	for (int blockX = 0; blockX < N; blockX++) {
		for (int blockY = 0; blockY < N; blockY++) {


			for (int i = blockX * n; i < (blockX + 1)*n; i++) {
				for (int j = blockY * n; j < (blockY + 1)*n; j++) {

					for (int ki = blockX * n; ki <= i; ki++) {
						for (int kj = blockY * n; kj <= j; kj++) {
							int c = A[ki * n * N + kj];
							B[i*n *N + j] += A[ki * n *N + kj];
						}
					}
					B[i*n *N + j] -= A[i*n*N + j];

				}
			}

		}
	}

	time_f = omp_get_wtime();
	printf("CPU: %f \n",
		(time_f - time_s));

}

__global__ void SumGPU(int *a, int *b, int N) {
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dx = blockDim.x;
	int dy = blockDim.y;
	int i = blockX * dx + tx;
	int j = blockY * dy + ty;

	for (int ki = blockX * n; ki <= i; ki++) {
		for (int kj = blockY * n; kj <= j; kj++) {
			int c = a[ki * n*N + kj];
			b[i*n *N + j] += a[ki * n*N + kj];
		}
	}
	b[i*n*N + j] -= a[i*n*N + j];
}



__global__ void SumGPU_2(int *a, int *b, int N) {
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dx = blockDim.x;
	int dy = blockDim.y;
	int j = blockX * dx + tx;
	int i = blockY * dy + ty;

	for (int ki = blockY * n; ki <= i; ki++) {
		for (int kj = blockX * n; kj <= j; kj++) {
			int c = a[ki * n*N + kj];
			b[i*n *N + j] += a[ki * n*N + kj];
		}
	}
	b[i*n*N + j] -= a[i*n*N + j];
}



__global__ void SumGPU_3(int *a, int *b, int N) {

	__shared__ int  smem[n][n];

	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dx = blockDim.x;
	int dy = blockDim.y;
	int j = blockX * dx + tx;
	int i = blockY * dy + ty;

	smem[ty][tx] = a[i*n *N + j];
	__syncthreads();

	for (int ki = 0; ki <= ty; ki++) {
		for (int kj = 0; kj <= tx; kj++) {
			b[i*n *N + j] += smem[ki][kj];
		}
	}
	b[i*n*N + j] -= smem[ty][tx];
	__syncthreads();
}


using namespace std;

void main(int argc, char* argv[])
{
	int count;
	//printf("count = ");
	//scanf("%d",&count);
	//int N = count % n == 0 ? count / n : count / n + 1;
	int N = 32;
	double time_s, time_f;

	int *A = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);
	int *B = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);


	for (int i = 0; i < n*N; i++)
	{
		for (int j = 0; j < n*N; j++)
		{
			A[i*n*N + j] = i * n*N + j;
		}
	}

	memcpy(B, A, n *n *N*N * sizeof(int));

	SumCPU(A, B, N);

	//for (int i = 0; i < n*N; i++)
	//{
	//	for (int j = 0; j < n*N; j++)
	//	{
	//		printf("%u\t", A[i*n*N + j]);
	//	}
	//	printf("\n");
	//}
	//printf("\n");
	//printf("\n");


	/*for (int i = 0; i < n*N; i++)
	{
	for (int j = 0; j < n*N; j++)
	{
	printf("%u\t", B[i*n*N + j]);
	}
	printf("\n");
	}

	printf("\n");
	printf("\n");*/

	int *dev_A, *dev_X, *dev_B, *dev_B2, *dev_B3;

	int *a = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);
	int *b = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);
	int *b_2 = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);
	int *b_3 = (int*)_aligned_malloc(n *n *N*N * sizeof(int), 32);

	memcpy(a, A, n *n *N*N * sizeof(int));
	memcpy(b, A, n *n *N*N * sizeof(int));
	memcpy(b_2, A, n *n *N*N * sizeof(int));
	memcpy(b_3, A, n *n *N*N * sizeof(int));

	hipError_t error = hipMalloc((void**)&dev_A, n *n *N*N * sizeof(int));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMalloc((void**)&dev_B, n *n *N*N * sizeof(int));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMalloc((void**)&dev_B2, n *n *N*N * sizeof(int));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMalloc((void**)&dev_B3, n *n *N*N * sizeof(int));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(dev_A, a, n *n *N*N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(dev_B, b, n *n *N*N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(dev_B2, b_2, n *n *N*N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(dev_B3, b_3, n *n *N*N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	dim3 threads(n, n);
	//dim3 blocks((n + (fiber - 1)) / fiber, (n + (fiber - 1)) / fiber);
	dim3 blocks(N, N);

	//begin = __rdtsc();
	time_s = omp_get_wtime();
	hipEventSynchronize(start);
	SumGPU << <blocks, threads >> >(dev_A, dev_B, N);
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//end = __rdtsc() - begin;

	//printf("runTimeGPU1 =  %llu \n", end - begin);
	time_f = omp_get_wtime();
	printf("GPU_1: %f \n",
		(time_f - time_s));

	error = hipMemcpy(b, dev_B, n*n *N*N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	float timer = 0;

	time_s = omp_get_wtime();
	hipEventSynchronize(start);
	SumGPU_2 << <blocks, threads >> >(dev_A, dev_B2, N);
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//end = __rdtsc() - begin;

	//printf("runTimeGPU1 =  %llu \n", end - begin);
	time_f = omp_get_wtime();
	printf("GPU_2: %f \n",
		(time_f - time_s));

	error = hipMemcpy(b_2, dev_B2, n*n *N*N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	timer = 0;

	time_s = omp_get_wtime();
	hipEventSynchronize(start);
	SumGPU_3 << <blocks, threads >> >(dev_A, dev_B3, N);
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//end = __rdtsc() - begin;

	//printf("runTimeGPU1 =  %llu \n", end - begin);
	time_f = omp_get_wtime();
	printf("GPU_3 with __shared__: %f \n",
		(time_f - time_s));

	error = hipMemcpy(b_3, dev_B3, n*n *N*N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	//hipEventElapsedTime(&timer, start, stop);
	//printf("GPU = %f ms\n", timer);


	//cout << "runTimeGPU = " << timer << endl;

	/*for (int i = 0; i < n*N; i++)
	{
	for (int j = 0; j < n*N; j++)
	{
	printf("%u\t", b[i*n*N + j]);
	}
	printf("\n");
	}
	printf("\n");
	printf("\n");
	*/
	int ch = 0;
	for (int i = 0; i < n*N; i++)
	{
		for (int j = 0; j < n*N; j++)
		{
			if (B[i*n*N + j] != b[i*n*N + j]) {
				printf("B %d - b %d\n", B[i*n*N + j], b[i*n*N + j]);
				printf("ERROR!!! b1\n");
				ch = 1;
			}

			if (B[i*n*N + j] != b_2[i*n*N + j]) {
				printf("B %d - b2 %d\n", B[i*n*N + j], b_2[i*n*N + j]);
				printf("ERROR!!! b2\n");
				ch = 1;
			}

			if (B[i*n*N + j] != b_3[i*n*N + j]) {
				printf("B %d - b3 %d\n", B[i*n*N + j], b_3[i*n*N + j]);
				printf("ERROR!!! b3\n");
				ch = 1;
			}

			if (ch == 1) {
				hipFree(dev_A);
				hipFree(dev_B);
				hipFree(dev_B2);
				hipFree(dev_B3);
				return;
			}
		}
	}

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_B2);
	hipFree(dev_B3);
}
