#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/opencv.hpp>

#include <conio.h>
#include <Windows.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <ctime>
#include <intrin.h>
#include <omp.h>

using namespace cv;
using namespace std;
const int nx = 32;
const int ny = 16;
const int N = 4;
const int rgb = 3;

__global__ void SumGPU(unsigned *a, char *b, unsigned char *r,		//
	int cols, int rows, int pitch, int pitch1) {

	const int ex = 1;
	__shared__ unsigned smem[ny + 2][(nx + ex)*rgb];

	int xid = (blockIdx.x * blockDim.x + threadIdx.x)*rgb;

	int yid = blockIdx.y * blockDim.y + threadIdx.y;

	//r[yid * pitch1 + xid] = a[yid * pitch + xid];

	if (yid >= rows) {
		return;
	}

	if ((threadIdx.y + 1) == ny)
	{
		smem[threadIdx.y][threadIdx.x*rgb] = a[yid * pitch + xid];
		smem[threadIdx.y + 1][threadIdx.x*rgb] = a[(yid + 1) * pitch + xid];
		smem[threadIdx.y + 2][threadIdx.x*rgb] = a[(yid + 2) * pitch + xid];

		smem[threadIdx.y][threadIdx.x*rgb+1] = a[yid * pitch + xid+1];
		smem[threadIdx.y + 1][threadIdx.x*rgb+1] = a[(yid + 1) * pitch + xid+1];
		smem[threadIdx.y + 2][threadIdx.x*rgb+1] = a[(yid + 2) * pitch + xid+1];

		smem[threadIdx.y][threadIdx.x*rgb+2] = a[yid * pitch + xid+2];
		smem[threadIdx.y + 1][threadIdx.x*rgb+2] = a[(yid + 1) * pitch + xid+2];
		smem[threadIdx.y + 2][threadIdx.x*rgb+2] = a[(yid + 2) * pitch + xid+2];

		if ((threadIdx.x + 1) == nx)
		{
			smem[threadIdx.y][(threadIdx.x + 1)*rgb] = a[yid * pitch + xid + 3];
			smem[threadIdx.y + 1][(threadIdx.x + 1) * rgb] = a[(yid + 1) * pitch + xid + 3];
			smem[threadIdx.y + 2][(threadIdx.x + 1) * rgb] = a[(yid + 2) * pitch + xid + 3];

			smem[threadIdx.y][(threadIdx.x + 1)*rgb+1] = a[yid * pitch + xid + 3+1];
			smem[threadIdx.y + 1][(threadIdx.x + 1) * rgb+1] = a[(yid + 1) * pitch + xid + 3+1];
			smem[threadIdx.y + 2][(threadIdx.x + 1) * rgb+1] = a[(yid + 2) * pitch + xid + 3+1];

			smem[threadIdx.y][(threadIdx.x + 1)*rgb+2] = a[yid * pitch + xid + 3+2];
			smem[threadIdx.y + 1][(threadIdx.x + 1) * rgb+2] = a[(yid + 1) * pitch + xid + 3+2];
			smem[threadIdx.y + 2][(threadIdx.x + 1) * rgb+2] = a[(yid + 2) * pitch + xid + 3+2];			
		}
	}
	else
	{
		smem[threadIdx.y][(threadIdx.x)*rgb] = a[yid * pitch + xid];
		smem[threadIdx.y][(threadIdx.x)*rgb+1] = a[yid * pitch + xid+1];
		smem[threadIdx.y][(threadIdx.x)*rgb+2] = a[yid * pitch + xid+2];

		if ((threadIdx.x + 1) == nx)
		{
			smem[threadIdx.y][(threadIdx.x + 1)*rgb] = a[yid * pitch + xid + 3];
			smem[threadIdx.y][(threadIdx.x + 1)*rgb+1] = a[yid * pitch + xid + 3+1];
			smem[threadIdx.y][(threadIdx.x + 1)*rgb+2] = a[yid * pitch + xid + 3+2];			
		}
	}
	
	__syncthreads();

	unsigned char *psmem = (unsigned char *)smem;

		for (int i = 0; i < 12 && (xid)* N + i + 3 < cols - 3
		&& yid + 1 < rows - 1; i++)
	{ 
		int basex = threadIdx.x * N*rgb + i;
		int basey = threadIdx.y;

		int sum = 0;

		// ������������ !!!!!!!!!!!!!!!!!!!!!!!!!!!!
		sum += psmem[basey*((nx + ex)*N*rgb) + basex] * b[0];
		sum += psmem[basey*((nx + ex)*N*rgb) + basex + 3] * b[1];
		sum += psmem[basey*((nx + ex)*N*rgb) + basex + 6] * b[2];
		sum += psmem[(basey + 1)*((nx + ex)*N*rgb) + basex] * b[3];
		sum += psmem[(basey + 1)*((nx + ex)*N*rgb) + basex + 3] * b[4];
		sum += psmem[(basey + 1)*((nx + ex)*N*rgb) + basex + 6] * b[5];
		sum += psmem[(basey + 2)*((nx + ex)*N*rgb) + basex] * b[6];
		sum += psmem[(basey + 2)*((nx + ex)*N*rgb) + basex + 3] * b[7];
		sum += psmem[(basey + 2)*((nx + ex)*N*rgb) + basex + 6] * b[8];

		if (sum > 255) {
			sum = 255;
		}
		if (sum < 0) {
			sum = 0;
		}

		// �������������� !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
		r[(yid+1)* pitch1 + (xid)* N + i + 3] = (unsigned char)sum;
	}
}



int main()
{
	Mat img = imread("D:\\test1.ppm", IMREAD_UNCHANGED);	
	normalize(img, img, 0, 255, NORM_MINMAX);
	img.convertTo(img, CV_8U);

	Mat imgRes = img.clone();
	char B[] = { -1,-1,-1,-1,9,-1,-1,-1,-1 };

	srand((unsigned int)time(NULL));
	double time_s, time_f;
	time_s = omp_get_wtime();

	for (int i = 1; i < (img.rows - 1); i++) {
		for (int j = 3; j < (img.cols - 1)*3; j++) {
			int basei = i - 1;
			int basej = j - 3;
			int sum = 0;
			for (int it = basei; it < basei + 3; it++) {
				for (int jt = basej; jt < basej + 9; jt+=3)
				{
					sum += img.data[it*img.cols * 3 + jt] * B[(it - basei) * 3 + (jt- basej)/3];
				}
			}

			if (sum > 255) {
				sum = 255;
			}
			if (sum < 0) {
				sum = 0;
			}
			imgRes.data[i*img.cols * 3 + j] = (unsigned char)sum;
		}
	}

	time_f = omp_get_wtime();
	printf("CPU: %f \n",
		(time_f - time_s));

	unsigned char *dev_R;		//
	unsigned *dev_A;
	char *dev_B;
	Mat imgResGPU = img.clone();

	int cols = img.cols*3;
	size_t pitch, pitch1;
	size_t host_orig_pitch = cols * sizeof(unsigned char);
	size_t host_orig_pitch_A = cols * sizeof(unsigned);

	hipError_t error = hipMallocPitch((void**)&dev_A, &pitch, cols * sizeof(unsigned), img.rows);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMallocPitch((void**)&dev_R, &pitch1, cols * sizeof(unsigned char), img.rows);		//
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipMemset(dev_R,0, pitch1*img.rows);

	error = hipMemcpy2D(dev_A, pitch, img.data, host_orig_pitch, cols * sizeof(unsigned char), img.rows, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy2D(dev_R, pitch1, img.data, host_orig_pitch, cols * sizeof(unsigned char), img.rows, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMalloc((void**)&dev_B, 9 * sizeof(char));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_B, B, 9 * sizeof(char), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}


	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	dim3 threads(nx, ny);

	int blockx = (cols + nx * N * rgb - 1) / (nx * N * rgb);  //pitch % nx*N == 0 ? pitch / (nx *N) : pitch / (nx*N) + 1;
	int blocky = (img.rows + ny - 1) / (ny);  //pitch % ny == 0 ? pitch / ny : pitch / ny + 1;

	dim3 blocks(blockx, blocky);

	time_s = omp_get_wtime();
	error = hipEventSynchronize(start);
	SumGPU << <blocks, threads >> > (dev_A, dev_B, dev_R, cols, img.rows, pitch/sizeof(unsigned), pitch1);		//
	error = hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	time_f = omp_get_wtime();
	printf("GPU_1: %f \n",
		(time_f - time_s));

	error = hipMemcpy2D(imgResGPU.data, host_orig_pitch, dev_R, pitch1, cols, imgResGPU.rows, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_R);

	printf("OK\n");
	imwrite("D:\\outputGPU.pgm", imgResGPU);
	imwrite("D:\\outputCPU.pgm", imgRes);
	//imwrite("D:\\outputImg.pgm", img);
	namedWindow("imgRes", CV_WINDOW_OPENGL);	
	imshow("imgRes", imgRes);
	//namedWindow("imgRes", CV_WINDOW_OPENGL);
	//imshow("imgRes", imgRes);
	namedWindow("imgResGPU", CV_WINDOW_OPENGL);
	imshow("imgResGPU", imgResGPU);
	waitKey(0);
	return 0;
}
