#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/opencv.hpp>

#include <conio.h>
#include <Windows.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <ctime>
#include <intrin.h>
#include <omp.h>

using namespace cv;
using namespace std;
const int nx = 32;
const int ny = 16;
const int N = 4;


__global__ void SumGPU(unsigned *a, char *b, unsigned char *r,
	int cols, int rows, int pitch, int pitch1) {

	__shared__ unsigned smem[ny + 1][nx + 1];
	unsigned tempBlock[12] = { 0,0,0,0,0,0,0,0,0,0,0,0 };

	int xid = blockIdx.x * blockDim.x + threadIdx.x;
	int yid = blockIdx.y * blockDim.y + threadIdx.y;

	if (blockIdx.x == (gridDim.x - 1) || blockIdx.y == (gridDim.y - 1))
		return;

	if (xid < cols && yid < rows); {

		if ((threadIdx.y + 1) == ny && (yid + 1) < rows)
		{
			smem[threadIdx.y][threadIdx.x] = a[yid * pitch + xid];
			smem[threadIdx.y + 1][threadIdx.x] = a[(yid + 1) * pitch + xid];

			if ((threadIdx.x + 1) == nx)
			{
				smem[threadIdx.y][(threadIdx.x + 1)] = a[yid * pitch + xid + 1];
				smem[threadIdx.y + 1][threadIdx.x + 1] = a[(yid + 1) * pitch + xid + 1];
				//printf("CPU: %u__ GPU: %u", a[yid * pitch + xid * N + e], smem[threadIdx.y][threadIdx.x * N + e]);
			}
		}
		else
		{
			smem[threadIdx.y][threadIdx.x] = a[yid * pitch + xid];

			if ((threadIdx.x + 1) == nx)
			{
				smem[threadIdx.y][(threadIdx.x + 1)] = a[yid * pitch + xid + 1];
				//printf("CPU: %u__ GPU: %u", a[yid * pitch + xid * N + e], smem[threadIdx.y][threadIdx.x * N + e]);
			}
		}
		__syncthreads();


		unsigned char *ptr = (unsigned char *)smem;
		if (yid < rows - 1 && xid < cols - 1)
		{
			int esx = 0, esy = 0;
			if (threadIdx.x == 0) {
				esx = 1;
			}
			if (threadIdx.y == 0) {
				esy = 1;
			}
			for (; esx < 4 && threadIdx.x * N + esx < cols; esx++)
			{
				int basex = threadIdx.x * N - 1 + esx;
				int basey = threadIdx.y - 1 + esy;
				int sum = 0;
				for (int yt = basey; yt < basey + 3; yt++) {
					for (int xt = basex; xt < basex + 3; xt++) {
						sum += ptr[yt*(nx)*N + 1 + xt] * b[(yt - basey) * 3 + (xt - basex)];
					}
				}
				if (sum > 255) {
					sum = 255;
				}
				if (sum < 0) {
					sum = 0;
				}
				r[yid * pitch1 + xid * N + esx] = (unsigned char)sum;
			}
		}

		//unsigned char *ptr = (unsigned char *)smem;
		//if (yid < rows - 1 && xid*N < cols - 1) 
		//{
		//	int esx = 0, esy = 0;
		//	if (threadIdx.x == 0) {
		//		esx = 1;				
		//	}
		//	if (threadIdx.y == 0) {
		//		esy = 1;
		//	}
		//	for (; esx < 4 && threadIdx.x * N + esx < cols; esx++)
		//	{
		//		int basex = threadIdx.x * N - 1 + esx;
		//		int basey = threadIdx.y - 1 + esy;
		//		int sum = 0;
		//		for (int yt = basey; yt < basey + 3; yt++) {
		//			for (int xt = basex; xt < basex + 3; xt++) {
		//				sum += ptr[yt*(nx)*N + 1 + xt] * b[(yt - basey) * 3 + (xt - basex)];
		//			}
		//		}
		//		if (sum > 255) {
		//			sum = 255;
		//		}
		//		if (sum < 0) {
		//			sum = 0;
		//		}
		//		r[yid * pitch1 + xid*N + esx] = (unsigned char)sum;
		//	}			
		//}		
	}

}

int main()
{
	Mat img = imread("D:\\test.pgm", IMREAD_UNCHANGED);
	normalize(img, img, 0, 255, NORM_MINMAX);
	img.convertTo(img, CV_8U);

	Mat imgRes = img.clone();
	unsigned char *A = (unsigned char*)_aligned_malloc(img.rows * img.cols * sizeof(unsigned char), 8);
	char B[] = { -1,-1,-1,-1,9,-1,-1,-1,-1 };

	srand((unsigned int)time(NULL));
	double time_s, time_f;
	time_s = omp_get_wtime();

	for (int i = 1; i < img.rows - 1; i++) {
		for (int j = 1; j < img.cols - 1; j++) {
			int basei = i - 1;
			int basej = j - 1;
			int sum = 0;
			for (int it = basei; it < basei + 3; it++) {
				for (int jt = basej; jt < basej + 3; jt++)
				{
					sum += img.data[it*img.cols + jt] * B[(it - basei) * 3 + (jt - basej)];
				}
			}

			if (sum > 255) {
				sum = 255;
			}
			if (sum < 0) {
				sum = 0;
			}
			imgRes.data[i*img.cols + j] = (unsigned char)sum;
		}
	}

	time_f = omp_get_wtime();
	printf("CPU: %f \n",
		(time_f - time_s));

	unsigned char *dev_R;
	unsigned *dev_A;
	char *dev_B;
	Mat imgResGPU = img.clone();

	size_t pitch, pitch1;
	size_t host_orig_pitch = img.cols * sizeof(unsigned char);
	size_t host_orig_pitch_A = img.cols * sizeof(unsigned);

	hipError_t error = hipMallocPitch((void**)&dev_A, &pitch, img.cols * sizeof(unsigned), img.rows);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMallocPitch((void**)&dev_R, &pitch1, img.cols * sizeof(unsigned char), img.rows);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy2D(dev_A, pitch, img.data, host_orig_pitch, img.cols * sizeof(unsigned char), img.rows, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy2D(dev_R, pitch1, img.data, host_orig_pitch, img.cols * sizeof(unsigned char), img.rows, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMalloc((void**)&dev_B, 9 * sizeof(char));
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_B, B, 9 * sizeof(char), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}


	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	dim3 threads(nx, ny);

	int blockx = (img.cols + nx * N - 1) / (nx * N);  //pitch % nx*N == 0 ? pitch / (nx *N) : pitch / (nx*N) + 1;
	int blocky = (img.rows + ny - 1) / (ny);  //pitch % ny == 0 ? pitch / ny : pitch / ny + 1;
	dim3 blocks(blockx, blocky);

	time_s = omp_get_wtime();
	error = hipEventSynchronize(start);
	SumGPU << <blocks, threads >> > (dev_A, dev_B, dev_R, img.cols, img.rows, pitch / sizeof(unsigned), pitch1);
	error = hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	time_f = omp_get_wtime();
	printf("GPU_1: %f \n",
		(time_f - time_s));

	error = hipMemcpy2D(imgResGPU.data, host_orig_pitch, dev_R, pitch1, imgResGPU.cols, imgResGPU.rows, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}



	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_R);

	printf("OK\n");
	imwrite("D:\\outputGPU.pgm", imgResGPU);
	imwrite("D:\\outputCPU.pgm", imgRes);
	imwrite("D:\\outputImg.pgm", img);
	//imshow("img", img);
	//imshow("imgRes", imgRes);
	//imshow("imgResGPU", imgResGPU);	
	waitKey(0);
	return 0;
}
